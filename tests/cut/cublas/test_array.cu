#include "hip/hip_runtime.h"
/**
  * @file test_array.cu
  * @brief Test the array operations in cuBLAS
  * @author sailing-innocent
  * @date 2025-01-18
  */

#include "test_util.h"

#include <hip/hip_runtime_api.h>
#include <vector>
#include <hipblas.h>
#include "SailCuT/dummy.h"

namespace sail::test {

using namespace sail::cu;

template<CommonValueType T>
int cublasIMax(hipblasHandle_t handle, int n, const T* x, int incx, int* result) {
	// switch (T)
	if constexpr (std::is_same_v<T, float>) {
		CUBLAS_CHECK(hipblasIsamax(handle, n, x, incx, result));
	} else if constexpr (std::is_same_v<T, double>) {
		CUBLAS_CHECK(hipblasIdamax(handle, n, x, incx, result));
	} else if constexpr (std::is_same_v<T, hipComplex>) {
		CUBLAS_CHECK(hipblasIcamax(handle, n, x, incx, result));
	} else if constexpr (std::is_same_v<T, hipDoubleComplex>) {
		CUBLAS_CHECK(hipblasIzamax(handle, n, x, incx, result));
	} else {
		throw std::runtime_error("Unsupported type");
	}
	return 0;
}

template<CommonValueType T>
bool test_cublas_array() {
	// using namespace sail::cu;
	hipblasHandle_t cublasH = nullptr;
	hipStream_t stream = nullptr;
	// A = | 1.0 2.0 3.0 4.0 |
	const std::vector<T> A = {T(1), T(2), T(3), T(4)};
	const int incx = 1;
	int result = 0.0;
	// step 1: create cublas handle, bind a stream
	CUBLAS_CHECK(hipblasCreate(&cublasH));
	CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

	// step 2: copy data to device
	T* d_A = nullptr;
	CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(T) * A.size()));
	CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(T) * A.size(), hipMemcpyHostToDevice, stream));

	// step 3: compute
	cublasIMax<T>(cublasH, A.size(), d_A, incx, &result);// return the max value index to a host pointer

	CUDA_CHECK(hipStreamSynchronize(stream));
	// result = 4
	CHECK(result == 4);

	// free resources
	CUDA_CHECK(hipFree(d_A));
	CUBLAS_CHECK(hipblasDestroy(cublasH));
	CUDA_CHECK(hipStreamDestroy(stream));
	CUDA_CHECK(hipDeviceReset());
	return true;
}

}// namespace sail::test

TEST_SUITE("cublas") {
	TEST_CASE("basic_array") {
		CHECK(sail::test::test_cublas_array<double>());
		CHECK(sail::test::test_cublas_array<float>());
	}
}