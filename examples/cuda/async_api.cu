#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void increment_kernel(int* g_data, int inc_value) {
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	g_data[idx] += inc_value;
}

int main(int argc, char** argv) {
	int devID;
	hipDeviceProp_t deviceProp;
}