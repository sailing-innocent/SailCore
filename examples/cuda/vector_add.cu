/**
  * @file vertex_add.cu
  * @brief The Basic Hello World Example to Learn NSight Profiler
  * @author sailing-innocent
  * @date 2025-01-18
  */


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define N 10000

__global__ void add(int* a, int* b, int* c) {
	int tid = blockIdx.x;
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

int main() {
	std::vector<int> a(N, 1);
	std::vector<int> b(N, 2);
	std::vector<int> c(N, 0);

	int* d_a;
	int* d_b;
	int* d_c;

	hipMalloc(&d_a, N * sizeof(int));
	hipMalloc(&d_b, N * sizeof(int));
	hipMalloc(&d_c, N * sizeof(int));

	hipMemcpy(d_a, a.data(), N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), N * sizeof(int), hipMemcpyHostToDevice);

	add<<<N, 1>>>(d_a, d_b, d_c);

	hipMemcpy(c.data(), d_c, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}